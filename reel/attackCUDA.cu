
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <string.h>

#define DICT_WORD_SIZE 25
#define DICT_LENGHT 22740
#define MAX_READ_SIZE 10000
#define NBCUDA 3

typedef char TITLES[MAX_READ_SIZE][DICT_WORD_SIZE];

//Define un HASH = 40 caractères = 5*DOUBLE de 8 octets
typedef struct {
	long long p1;
	long long p2;
	long long p3;
	long long p4;
	long long p5;
} HASH;

//Check if two HASH is the same
__device__ bool same_hash(HASH* one, HASH* two){
	return one->p1==two->p1 &&
	       one->p2==two->p2 &&
	       one->p3==two->p3 &&
	       one->p4==two->p4 &&
	       one->p5==two->p5;
}

// Two fonctions which help manage files
// --------
FILE* openFile(char* url){
	FILE * ds = fopen(url, "r");
	if (ds == NULL){
		exit(EXIT_FAILURE);
	}
	return ds;
}

char * line = NULL;
size_t len = 0;
char * readline(FILE * f){
    ssize_t read;
    if ((read = getline(&line, &len, f)) != -1) {
	line[read-1] = '\0';
      return line;
    }
	return NULL;
}
// --------

//Lets define the CUDA fonction
__global__ void gpu(HASH* sha_tab, HASH* hash_tab, char* result){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int stride = blockDim.x * gridDim.x;
	for (int i = x; i < DICT_LENGHT; i += stride){
		for (int j = y; j < MAX_READ_SIZE; j += stride){
			if(same_hash(&sha_tab[i], &hash_tab[j])){
				result[j]=1;
			}
		}
	}
}


// The crack executable take the dict file and the hash in parameter
int main(int argc, char *argv[]) {
	char* dict_file = argv[1];
	char* sha_file = argv[2];

	HASH* sha_tab; hipMallocManaged(&sha_tab, sizeof(HASH)*DICT_LENGHT);
	FILE* ds = openFile(sha_file);
	char* currline = readline(ds);
	int nbLine = 0;
	while (currline!=NULL)
	{
		memcpy(&sha_tab[nbLine], currline, sizeof(HASH));
		nbLine++;

		currline = readline(ds);
	}
	fclose(ds);
	

	ds = openFile(dict_file);	
	for(int i=0;i<52;i++){
		//Store each line in an array
		TITLES* title_tab = (TITLES*)malloc(sizeof(TITLES));
		HASH* hash_tab; hipMallocManaged(&hash_tab, sizeof(HASH)*MAX_READ_SIZE);
		char* result; hipMallocManaged(&result, sizeof(char)*MAX_READ_SIZE); memset(result, 0, sizeof(char)*MAX_READ_SIZE);
		for(int j=0;j<MAX_READ_SIZE;j++){
			currline = readline(ds);
			char *tmp = strtok(currline, "\t");
			strcpy((*title_tab)[j], tmp);
			tmp = strtok(NULL, "\t");
			memcpy(&hash_tab[j], tmp, sizeof(HASH));
		}

		//Start the GPU
		dim3 dimBlock(NBCUDA, NBCUDA);
		dim3 dimGrid(NBCUDA, NBCUDA);
		gpu<<<dimGrid, dimBlock>>>(sha_tab, hash_tab, result);
		hipDeviceSynchronize();

		//Print the final result
		for(int i=0;i<MAX_READ_SIZE;i++){
			if(result[i]==1) printf("FINDED - %s\n", (*title_tab)[i]);
		}

		hipFree(title_tab);
		hipFree(hash_tab);
	}

	fclose(ds);	
    return 0;
}
