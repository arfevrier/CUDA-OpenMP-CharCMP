
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <string.h>

#define DICT_WORD_SIZE 25
#define DICT_LENGHT 22740

typedef char TITLES[DICT_LENGHT][DICT_WORD_SIZE];

//Define un HASH = 40 caractères = 5*DOUBLE de 8 octets
typedef struct {
	double p1;
	double p2;
	double p3;
	double p4;
	double p5;
} HASH;

//Check if two HASH is the same
__device__ int same_hash(HASH* one, HASH* two){
	return one->p1==two->p1 &&
	       one->p2==two->p2 &&
	       one->p3==two->p3 &&
	       one->p4==two->p4 &&
	       one->p5==two->p5;
}

// Two fonctions which help manage files
// --------
FILE* openFile(char* url){
	FILE * ds = fopen(url, "r");
	if (ds == NULL){
		exit(EXIT_FAILURE);
	}
	return ds;
}

char * line = NULL;
size_t len = 0;
char * readline(FILE * f){
    ssize_t read;
    if ((read = getline(&line, &len, f)) != -1) {
	line[read-1] = '\0';
      return line;
    }
	return NULL;
}
// --------

//Lets define the CUDA fonction
__global__ void gpu(HASH* shadow_tab, HASH* hash_tab, char* result){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int stride = blockDim.x * gridDim.x;
	for (int i = x; i < DICT_LENGHT; i += stride){
		for (int j = y; j < DICT_LENGHT; j += stride){
			if(same_hash(&shadow_tab[i], &hash_tab[j])){
				result[i]=1;
			}
		}
	}
}


// The crack executable take the dict file and the hash in parameter
int main(int argc, char *argv[]) {
	char* dict_file = argv[1];
	char* sha_file = argv[2];
	
	//Store each line in an array
	TITLES* title_tab = (TITLES*)malloc(sizeof(TITLES));
	HASH* hash_tab; hipMallocManaged(&hash_tab, sizeof(HASH)*DICT_LENGHT);
	char* result; hipMallocManaged(&result, sizeof(char)*DICT_LENGHT); memset(result, 0, sizeof(char)*DICT_LENGHT);
	
	FILE * ds = openFile(dict_file);
	char *currline = readline(ds);
	int nbLine = 0;
	while (currline!=NULL)
	{
		char *tmp = strtok(currline, "\t");
		strcpy((*title_tab)[nbLine], tmp);
		tmp = strtok(NULL, "\t");
		memcpy(&hash_tab[nbLine], tmp, sizeof(HASH));
		//printf("%s-%s\n", title_tab+(nbLine*DICT_WORD_SIZE), hash_tab+(nbLine*DICT_HASH_SIZE));
		nbLine++;

		currline = readline(ds);
	}
	fclose(ds);
	
	//Store each hash to find in an array
	HASH* shadow_tab; hipMallocManaged(&shadow_tab, sizeof(HASH)*DICT_LENGHT);
	ds = openFile(sha_file);
	currline = readline(ds);
	nbLine = 0;
	while (currline!=NULL)
	{
		memcpy(&shadow_tab[nbLine], currline, sizeof(HASH));	
		nbLine++;		

		currline = readline(ds);
	}

	//Start the GPU
	dim3 dimBlock(32, 32);
	dim3 dimGrid(32, 32);
	gpu<<<dimGrid, dimBlock>>>(shadow_tab, hash_tab, result);
	hipDeviceSynchronize();

	//Print the final result
	for(int i=0;i<22740;i++){
		if(result[i]==1) printf("FINDED - %s\n", (*title_tab)[i]);
	}

	fclose(ds);	
    return 0;
}
