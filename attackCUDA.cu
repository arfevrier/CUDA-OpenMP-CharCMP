
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <string.h>

#define DICT_WORD_SIZE 25
#define DICT_LENGHT 22740

typedef char TITLES[DICT_LENGHT][DICT_WORD_SIZE];

//Define un HASH = 40 caractères = 5*DOUBLE de 8 octets
typedef struct {
	double p1;
	double p2;
	double p3;
	double p4;
	double p5;
} HASH;

//Check if two HASH is the same
__device__ int same_hash(HASH* one, HASH* two){
	return one->p1==two->p1 &&
	       one->p2==two->p2 &&
	       one->p3==two->p3 &&
	       one->p4==two->p4 &&
	       one->p5==two->p5;
}

// Two fonctions which help manage files
// --------
FILE* openFile(char* url){
	FILE * ds = fopen(url, "r");
	if (ds == NULL){
		exit(EXIT_FAILURE);
	}
	return ds;
}

char * line = NULL;
size_t len = 0;
char * readline(FILE * f){
    ssize_t read;
    if ((read = getline(&line, &len, f)) != -1) {
	line[read-1] = '\0';
      return line;
    }
	return NULL;
}
// --------

//Lets define the CUDA fonction
__global__ void gpu(HASH* shadow_tab, HASH* hash_tab, TITLES* title_tab){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int stride = blockDim.x * gridDim.x;
	int i, j;
	for (i = x; i < 22740; i += stride){
		for (j = y; j < 22740; j += stride){
			if(same_hash(&shadow_tab[i], &hash_tab[j])){
				printf("FINDED - %s\n", (*title_tab)[j]);
			}
		}
	}
}


// The crack executable take the dict file and the hash in parameter
// ./crack dict.txt shadow.txt
int main(int argc, char *argv[]) {
	char* dict_file = argv[1];
	char* sha_file = argv[2];

	// Convert the hash to HASH type
	//for (size_t count = 0; count < sizeof val/sizeof *val; count++) {
	//	sscanf(sha_hash, "%2hhx", &val[count]);
	//	SHAtoFind += 2;
	//}
	
	//Store each line in an array
	TITLES* title_tab; hipMallocManaged(&title_tab, sizeof(TITLES));
	HASH* hash_tab; hipMallocManaged(&hash_tab, sizeof(HASH)*22740);
	
	FILE * ds = openFile(dict_file);
	char *currline = readline(ds);
	int nbLine = 0;
	while (currline!=NULL)
	{
		char *tmp = strtok(currline, " ");
		strcpy((*title_tab)[nbLine], tmp);
		tmp = strtok(NULL, " ");
		memcpy(&hash_tab[nbLine], tmp, sizeof(HASH));
		//printf("%s-%s\n", title_tab+(nbLine*DICT_WORD_SIZE), hash_tab+(nbLine*DICT_HASH_SIZE));
		nbLine++;

		currline = readline(ds);
	}
	fclose(ds);
	
	//Store each hash to find in an array
	HASH* shadow_tab; hipMallocManaged(&shadow_tab, sizeof(HASH)*22740);
	ds = openFile(sha_file);
	currline = readline(ds);
	nbLine = 0;
	while (currline!=NULL)
	{
		memcpy(&shadow_tab[nbLine], currline, sizeof(HASH));	
		nbLine++;		

		currline = readline(ds);
	}

	//Start the GPU
	dim3 dimBlock(16, 16);
	dim3 dimGrid(16, 16);
	gpu<<<dimGrid, dimBlock>>>(shadow_tab, hash_tab, title_tab);
	hipDeviceSynchronize();
	fclose(ds);
	
    return 0;
}
